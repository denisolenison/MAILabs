#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <chrono>

#include <windows.h>


using namespace std;

#define FIND_ERR(call) { gpuAssert((call), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t cudaStatus, const char* file, int line, bool abort = true)
{
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "ERROR: CUDA failed in %s:%d: %s\n", file, line, hipGetErrorString(cudaStatus));
		exit(0);
	}
}

__global__ void findMin(int N, double* v1, double* v2) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	while (tId < N) {
		if (v1[tId] < v2[tId]) {
			v2[tId] = v1[tId];
		}
		tId += blockDim.x * gridDim.x;
	}
}

//cpu
void findMinCPU(int N, double* v1, double* v2) {
	for (int i = 0 ; i < N ; ++i) {
		v2[i] = min(v1[i], v2[i]);
	}
}

int main()
{
	std::ios::sync_with_stdio(false);

	int N;
	cin >> N;
	if (N <= 0) {
		return 0;
	}

	double* vector1 = new double[N];
	double* vector2 = new double[N];

	double* fastV1, *fastV2;


	// �������������� ������ �� GPU
	FIND_ERR(hipMalloc(&fastV1, N * sizeof(double)));
	FIND_ERR(hipMalloc(&fastV2, N * sizeof(double)));

	for (int i = 0; i < N; i++) {
		vector1[i] = i;
	}

	for (int i = 0; i < N; i++) {
		vector2[i] = (i*i)%N;
	}


	//CPU FUNCT
	//������ �������
	/*
	LARGE_INTEGER perfCntStart, perfCntStop, proc_freq;
	::memset(&proc_freq, 0x00, sizeof(proc_freq));
	::memset(&perfCntStart, 0x00, sizeof(perfCntStart));
	::memset(&perfCntStop, 0x00, sizeof(perfCntStop));
	::QueryPerformanceCounter(&perfCntStart);
	::QueryPerformanceFrequency(&proc_freq);

	findMinCPU(N, vector1, vector2);
	
	::QueryPerformanceCounter(&perfCntStop);
	printf(": %f\n", float(perfCntStop.QuadPart - perfCntStart.QuadPart) / float(proc_freq.QuadPart) * 1000);
	*/


	//�������� � ������ GPU
	FIND_ERR(hipMemcpy(fastV1, vector1, N * sizeof(double), hipMemcpyHostToDevice));
	FIND_ERR(hipMemcpy(fastV2, vector2, N * sizeof(double), hipMemcpyHostToDevice));

	//������ ������� 

	hipEvent_t start, end;
	FIND_ERR(hipEventCreate(&start));
	FIND_ERR(hipEventCreate(&end));
	FIND_ERR(hipEventRecord(start));
	

	findMin <<<8, 8>>> (N, fastV1, fastV2); //�������
	FIND_ERR(hipGetLastError()); //�������� ������
	
	FIND_ERR(hipEventRecord(end));
	FIND_ERR(hipEventSynchronize(end));
	float t;
	FIND_ERR(hipEventElapsedTime(&t, start, end));
	FIND_ERR(hipEventDestroy(start));
	FIND_ERR(hipEventDestroy(end));
	//����� ������

	printf("time = %f\n", t);

	// �������� ��������� � ���
	FIND_ERR(hipMemcpy(vector2, fastV2, N * sizeof(double), hipMemcpyDeviceToHost));
	/*
	for (int i = 0; i < N; i++) {
		printf("%.10e ", vector2[i]);
	}*/

	// ������������ ������
	FIND_ERR(hipFree(fastV1));
	FIND_ERR(hipFree(fastV2));
	

	delete[] vector1;
	delete[] vector2;


    return 0;
}
