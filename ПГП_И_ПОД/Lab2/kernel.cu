#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <sstream>
#include <iomanip>
#include <math.h>
#include <algorithm>
#include <string>
#include <hip/hip_runtime.h>


using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}															    \
} while (0)		                                                    \
											                        \

typedef uchar4 pixels;
typedef unsigned char bytes;

struct image {
	int width;
	int height;
	pixels* pixs;
};

image newImage(int w, int h) {
	image nIMG;
	nIMG.width = w;
	nIMG.height = h;
	nIMG.pixs = new pixels[w * h];
	return nIMG;
}

image newImage(string filename) {
	FILE* file;
	image thisImg;
	if ((file = fopen(filename.c_str(), "rb")) == NULL) {
		std::cout << "Can't load image from file" << std::endl;
		exit(1);
	}

	fread(&thisImg.width, sizeof(thisImg.width), 1, file);
	fread(&thisImg.height, sizeof(thisImg.height), 1, file);

	thisImg.pixs = new pixels[thisImg.width * thisImg.height];
	fread(thisImg.pixs, sizeof(pixels), thisImg.width * thisImg.height, file);

	fclose(file);
	return thisImg;
}

void writeToFile(image img, string filename) {
	FILE* file = fopen(filename.c_str(), "wb");

	fwrite(&img.width, sizeof(img.width), 1, file);
	fwrite(&img.height, sizeof(img.height), 1, file);
	fwrite(img.pixs, sizeof(pixels), img.width * img.height, file);
	fclose(file);
}

string imgToString(image img) {
	std::stringstream stream;
	stream << img.width << " " << img.height << "\n";
	for (int i = 0; i < img.height; i++) {
		for (int j = 0; j < img.width; j++) {
			int k = i * img.width + j;
			stream << hex << setfill('0') << setw(2) << (int)img.pixs[k].x << setfill('0') << setw(2) << (int)img.pixs[k].y << setfill('0') << setw(2) << (int)img.pixs[k].z << setfill('0') << setw(2) << (int)img.pixs[k].w << " ";
		}
		stream << "\n";
	}

	return stream.str();
}

texture<pixels, 2, hipReadModeElementType> tex;

hipArray* c_arr;


void makeTexture(image* img) {
	int w = img->width;
	int h = img->height;

	hipChannelFormatDesc ch = hipCreateChannelDesc<pixels>();

	CSC(hipMallocArray(&c_arr, &ch, w, h));
	CSC(hipMemcpyToArray(c_arr, 0, 0, img->pixs, sizeof(pixels) * w * h, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;
	CSC(hipBindTextureToArray(tex, c_arr, tex.channelDesc));
}

__global__ void filterSSAA(pixels* pixelsOut, int w, int h, int psx, int psy)
{
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	int imW = w * psx;

	for (int i = tY; i < h; i += offsetY)
	{
		for (int j = tX; j < w; j += offsetX)
		{
			pixels thisPixel;
			double thisRed = 0.0;
			double thisGreen = 0.0;
			double thisBlue = 0.0;

			for (int Y = psy * i; Y < psy * i + psy; ++Y) {
				for (int X = psx * j; X < psx * j + psx; ++X) {
					thisPixel = tex2D(tex, X, Y);
					thisRed += thisPixel.x;
					thisGreen += thisPixel.y;
					thisBlue += thisPixel.z;
				}
			}

			thisRed /= (psx * psy);
			thisGreen /= (psx * psy);
			thisBlue /= (psx * psy);


			bytes nRed = (bytes)min((int)thisRed, (int)0xFF);
			bytes nGreen = (bytes)min((int)thisGreen, (int)0xFF);
			bytes nBlue = (bytes)min((int)thisBlue, (int)0xFF);


			pixelsOut[j + i * w].x = nRed;
			pixelsOut[j + i * w].y = nGreen;
			pixelsOut[j + i * w].z = nBlue;
			pixelsOut[j + i * w].w = 0;
		}
	}
}

void begin(image* image1, image* image2, int psX, int psY) {

	pixels* newPixels;

	makeTexture(image1);

	int size2 = sizeof(pixels) * image2->width * image2->height;

	CSC(hipMalloc((void**)& newPixels, size2));

	dim3 gridSz(32, 32);
	dim3 blockSz(32, 32);

	filterSSAA << < gridSz, blockSz >> > (newPixels, image2->width, image2->height, psX, psY);

	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(c_arr));

	CSC(hipMemcpy(image2->pixs, newPixels, size2, hipMemcpyDeviceToHost));
	CSC(hipFree(newPixels));

}

int main()
{
	string input;
	string output;

	int wNew, hNew;

	std::cin >> input >> output;
	std::cin >> wNew >> hNew;

	image myImage = newImage(input);

	int PS_x = 0;
	int PS_y = 0;

	if (myImage.width % wNew != 0 || myImage.height % hNew) {
		cout << "ERROR: Not prorortional!\n";
		return 0;
	}
	else {
		PS_x = myImage.width / wNew;
		PS_y = myImage.height / hNew;
	}

	image newIM = newImage(myImage.width / PS_x, myImage.height / PS_y);

	begin(&myImage, &newIM, PS_x, PS_y);

	writeToFile(newIM, output);

	return 0;
}