#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <sstream>
#include <iomanip>
#include <math.h>
#include <algorithm>
#include <string>
#include <hip/hip_runtime.h>


using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}															    \
} while (0)		                                                    \
											                        \

typedef uchar4 pixels;
typedef double pixFloat[3];
typedef pixFloat matrix3[3];


__constant__ pixFloat dev_cAvg[32];
__constant__ matrix3 dev_cMatrInv[32];

typedef unsigned char bytes;

struct image {
	int width;
	int height;
	pixels* pixs;
};

struct pixel {
	int x;
	int y;
};

image newImage(int w, int h) {
	image nIMG;
	nIMG.width = w;
	nIMG.height = h;
	nIMG.pixs = new pixels[w * h];
	return nIMG;
}

image newImage(string filename) {
	FILE* file;
	image thisImg;
	if ((file = fopen(filename.c_str(), "rb")) == NULL) {
		std::cout << "Can't load image from file" << std::endl;
		exit(1);
	}

	fread(&thisImg.width, sizeof(thisImg.width), 1, file);
	fread(&thisImg.height, sizeof(thisImg.height), 1, file);

	thisImg.pixs = new pixels[thisImg.width * thisImg.height];
	fread(thisImg.pixs, sizeof(pixels), thisImg.width * thisImg.height, file);

	fclose(file);
	return thisImg;
}

void writeToFile(image img, string filename) {
	FILE* file = fopen(filename.c_str(), "wb");

	fwrite(&img.width, sizeof(img.width), 1, file);
	fwrite(&img.height, sizeof(img.height), 1, file);
	fwrite(img.pixs, sizeof(pixels), img.width * img.height, file);
	fclose(file);
}

string imgToString(image img) {
	std::stringstream stream;
	stream << img.width << " " << img.height << "\n";
	for (int i = 0; i < img.height; i++) {
		for (int j = 0; j < img.width; j++) {
			int k = i * img.width + j;
			stream << hex << setfill('0') << setw(2) << (int)img.pixs[k].x << setfill('0') << setw(2) << (int)img.pixs[k].y << setfill('0') << setw(2) << (int)img.pixs[k].z << setfill('0') << setw(2) << (int)img.pixs[k].w << " ";
		}
		stream << "\n";
	}

	return stream.str();
}

__global__ void Mahalanobisse(pixels* pixelsOut, int w, int h, int classes)
{
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;

	for (int i = tY; i < h; i += offsetY)
	{
		for (int j = tX; j < w; j += offsetX)
		{
			pixels thisPixel = pixelsOut[j + i * w];
			double thisRed = (double)thisPixel.x;
			double thisGreen = (double)thisPixel.y;
			double thisBlue = (double)thisPixel.z;


			double maxAm = 0;
			int argMax = -1;
			for (int c = 0; c < classes; ++c) {
				double ans = 0;
				pixFloat vec1;
				pixFloat vec2;
				pixFloat vec3;

				vec1[0] = -(thisRed - dev_cAvg[c][0]);
				vec2[0] = thisRed - dev_cAvg[c][0];

				vec1[1] = -(thisGreen - dev_cAvg[c][1]);
				vec2[1] = thisGreen - dev_cAvg[c][1];

				vec1[2] = -(thisBlue - dev_cAvg[c][2]);
				vec2[2] = thisBlue - dev_cAvg[c][2];


				vec3[0] = vec1[0] * dev_cMatrInv[c][0][0] + vec1[1] * dev_cMatrInv[c][1][0] + vec1[2] * dev_cMatrInv[c][2][0];
				vec3[1] = vec1[0] * dev_cMatrInv[c][0][1] + vec1[1] * dev_cMatrInv[c][1][1] + vec1[2] * dev_cMatrInv[c][2][1];
				vec3[2] = vec1[0] * dev_cMatrInv[c][0][2] + vec1[1] * dev_cMatrInv[c][1][2] + vec1[2] * dev_cMatrInv[c][2][2];

				ans = vec3[0] * vec2[0] + vec3[1] * vec2[1] + vec3[2] * vec2[2];

				if (ans > maxAm || argMax == -1) {
					maxAm = ans;
					argMax = c;
				}

			}

			pixelsOut[j + i * w].w = argMax;

			//coloring 3 for report
			//if (argMax == 0) {
			//	pixelsOut[j + i * w].x = 255;
			//	pixelsOut[j + i * w].y = 0;
			//	pixelsOut[j + i * w].z = 0;
			//}
			//else if (argMax == 1) {
			//	pixelsOut[j + i * w].x = 0;
			//	pixelsOut[j + i * w].y = 255;
			//	pixelsOut[j + i * w].z = 0;
			//}
			//else if (argMax == 2) {
			//	pixelsOut[j + i * w].x = 0;
			//	pixelsOut[j + i * w].y = 0;
			//	pixelsOut[j + i * w].z = 255;
			//}

		}
	}
}

void begin(image* image1, int classes) {

	pixels* oldPixels;

	int size1 = sizeof(pixels) * image1->width * image1->height;

	CSC(hipMalloc((void**)& oldPixels, size1));

	int SIZE = 32;

	dim3 gridSz(SIZE, SIZE);
	dim3 blockSz(SIZE, SIZE);

	CSC(hipMemcpy(oldPixels, image1->pixs, size1, hipMemcpyHostToDevice));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	Mahalanobisse << < gridSz, blockSz >> > (oldPixels, image1->width, image1->height, classes);

	CSC(hipGetLastError()); //�������� ������

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));
	//����� ������
	printf("time = %f\n", t);


	CSC(hipMemcpy(image1->pixs, oldPixels, size1, hipMemcpyDeviceToHost));
	CSC(hipFree(oldPixels));

}





int main()
{
	string input;
	string output;

	int w;

	cin >> input >> output;

	image myImage = newImage(input);
	w = myImage.width;

	int classes;
	cin >> classes;

	double curRed = 0;
	double curGreen = 0;
	double curBlue = 0;

	pixFloat* cAvg = new pixFloat[classes];
	matrix3* cMatr = new matrix3[classes];
	matrix3* cMatrInv = new matrix3[classes];

	//��������� ������� �������� � ���. ������
	for (int i = 0; i < classes; ++i) {
		long long pixs_am = 0;

		curRed = 0;
		curGreen = 0;
		curBlue = 0;

		cin >> pixs_am;
		pixel* pixPairs = new pixel[pixs_am];
		for (long long j = 0; j < pixs_am; ++j) {
			int X, Y;
			cin >> X >> Y;
			curRed += (double)myImage.pixs[X + w * Y].x;
			curGreen += (double)myImage.pixs[X + w * Y].y;
			curBlue += (double)myImage.pixs[X + w * Y].z;
			pixPairs[j].x = X;
			pixPairs[j].y = Y;
		}
		curRed /= pixs_am;
		curGreen /= pixs_am;
		curBlue /= pixs_am;

		cAvg[i][0] = curRed;
		cAvg[i][1] = curGreen;
		cAvg[i][2] = curBlue;


		matrix3 totalMatrix;

		for (int Ti = 0; Ti < 3; ++Ti) {
			for (int Tj = 0; Tj < 3; ++Tj) {
				totalMatrix[Ti][Tj] = 0;
			}
		}

		for (int j = 0; j < pixs_am; ++j) {
			pixFloat vec;
			vec[0] = (double)myImage.pixs[pixPairs[j].x + w * pixPairs[j].y].x - cAvg[i][0];
			vec[1] = (double)myImage.pixs[pixPairs[j].x + w * pixPairs[j].y].y - cAvg[i][1];
			vec[2] = (double)myImage.pixs[pixPairs[j].x + w * pixPairs[j].y].z - cAvg[i][2];

			for (int Ti = 0; Ti < 3; ++Ti) {
				for (int Tj = 0; Tj < 3; ++Tj) {
					totalMatrix[Ti][Tj] += vec[Ti] * vec[Tj];
				}
			}
		}

		for (int Ti = 0; Ti < 3; ++Ti) {
			for (int Tj = 0; Tj < 3; ++Tj) {
				totalMatrix[Ti][Tj] /= max(0.000001, (double)pixs_am - 1);
				cMatr[i][Ti][Tj] = totalMatrix[Ti][Tj];
			}
		}

		delete[] pixPairs;

	}

	for (int i = 0; i < classes; ++i) {
		double det = 0;
		det = cMatr[i][0][0] * cMatr[i][1][1] * cMatr[i][2][2] + cMatr[i][0][2] * cMatr[i][1][0] * cMatr[i][2][1] +
			cMatr[i][0][1] * cMatr[i][1][2] * cMatr[i][2][0] - cMatr[i][2][0] * cMatr[i][1][1] * cMatr[i][0][2] -
			cMatr[i][0][1] * cMatr[i][1][0] * cMatr[i][2][2] - cMatr[i][0][0] * cMatr[i][1][2] * cMatr[i][2][1];
		if (det == 0) det = 0.0000001; //����� ��������� �� ��������
		matrix3 transp;
		for (int x = 0; x < 3; ++x) {
			for (int y = 0; y < 3; ++y) {
				transp[x][y] = cMatr[i][y][x];
			}
		}


		double dop1 = transp[1][1] * transp[2][2] - transp[1][2] * transp[2][1];
		double dop4 = transp[1][2] * transp[2][0] - transp[1][0] * transp[2][2];
		double dop7 = transp[1][0] * transp[2][1] - transp[1][1] * transp[2][0];

		double dop2 = transp[0][2] * transp[2][1] - transp[0][1] * transp[2][2];
		double dop5 = transp[0][0] * transp[2][2] - transp[0][2] * transp[2][0];
		double dop8 = transp[0][1] * transp[2][0] - transp[0][0] * transp[2][1];

		double dop3 = transp[0][1] * transp[1][2] - transp[0][2] * transp[1][1];
		double dop6 = transp[0][2] * transp[1][0] - transp[0][0] * transp[1][2];
		double dop9 = transp[0][0] * transp[1][1] - transp[0][1] * transp[1][0];

		cMatrInv[i][0][0] = dop1 / det;
		cMatrInv[i][0][1] = dop2 / det;
		cMatrInv[i][0][2] = dop3 / det;

		cMatrInv[i][1][0] = dop4 / det;
		cMatrInv[i][1][1] = dop5 / det;
		cMatrInv[i][1][2] = dop6 / det;

		cMatrInv[i][2][0] = dop7 / det;
		cMatrInv[i][2][1] = dop8 / det;
		cMatrInv[i][2][2] = dop9 / det;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(dev_cAvg), cAvg, sizeof(pixFloat) * classes, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(dev_cMatrInv), cMatrInv, sizeof(matrix3) * classes, 0, hipMemcpyHostToDevice);

	delete[] cAvg;
	delete[] cMatr;
	delete[] cMatrInv;


	begin(&myImage, classes);

	writeToFile(myImage, output);

	return 0;
}