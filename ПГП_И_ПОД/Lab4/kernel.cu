#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/swap.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <sstream>
#include <iomanip>
#include <math.h>
#include <algorithm>
#include <string>

using namespace std;
using namespace thrust;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}															    \
} while (0)	



const int BLOCK = 256;
const int THREAD = 256;

const int BLOCK2 = 16;
const int THREAD2 = 16;


__constant__ int N[1];
__constant__ int M[1];
__constant__ int K[1];

struct abs_fun : public thrust::unary_function<double, double> {
	__host__ __device__
		double operator()(double elem) const {
		return elem < 0 ? -elem : elem;
	}
};

struct abs_comp {
	abs_fun fabs;
	__host__ __device__ double operator()(double a, double b) {
		return fabs(a) < fabs(b);
	}
};



__global__ void fastSwap(int row, int col, int max_id, double* AB) {
	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned ts = blockDim.x * gridDim.x;

	for (int el = tx; el < M[0] + K[0]; el += ts) {
		double temp = AB[el * N[0] + row];
		AB[el * N[0] + row] = AB[el * N[0] + max_id];
		AB[el * N[0] + max_id] = temp;
	}
}

__global__ void triagStep(double* AB, int r, int c) {
	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned txs = blockDim.x * gridDim.x;

	unsigned ty = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned tys = blockDim.y * gridDim.y;

	for (int col_ = ty + c + 1; col_ < M[0] + K[0]; col_ += tys) {
		for (int row_ = tx + r + 1; row_ < N[0]; row_ += txs) {

			double rowDivisor = AB[row_ + N[0] * c] / AB[r + N[0] * c];
			AB[row_ + N[0] * col_] -= AB[r + N[0] * col_] * rowDivisor;

		}
	}
}



__global__ void backStep(double* AB, int row, int col) {
	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned txs = blockDim.x * gridDim.x;

	unsigned ty = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned tys = blockDim.y * gridDim.y;

	for (int col_ = ty + col + 1; col_ < M[0] + K[0]; col_ += tys) {
		for (int row_ = tx; row_ < row; row_ += txs) {
			double rowDivisor = AB[row_ + col * N[0]] / AB[row + col * N[0]];
			AB[row_ + col_ * N[0]] -= AB[row + col_ * N[0]] * rowDivisor;
		}
	}
}


__global__ void backKill(double* AB, int row, int col) {
	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned txs = blockDim.x * gridDim.x;

	for (int row_ = tx; row_ < row; row_ += txs) {
		double rowDivisor = AB[row_ + col * N[0]] / AB[row + col * N[0]];
		AB[row_ + col * N[0]] -= AB[row + col * N[0]] * rowDivisor;
	}


}

__global__ void division(double* AB, double* diag, int* indices, int* indices2, int freeX) {

	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned txs = blockDim.x * gridDim.x;

	unsigned ty = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned tys = blockDim.y * gridDim.y;


	for (int col_ = ty; col_ < K[0]; col_ += tys) {
		for (int i = tx; i < freeX; i += txs) {
			int curRow = indices2[i];

			double rowDivisor = diag[i];
			AB[curRow + col_ * N[0] + N[0] * M[0]] = AB[curRow + col_ * N[0] + N[0] * M[0]] / rowDivisor;

		}
	}
}




__global__ void triagKill(double* AB, int r, int c) {
	unsigned tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned txs = blockDim.x * gridDim.x;

	for (int row_ = tx + r + 1; row_ < N[0]; row_ += txs) {

		double rowDivisor = AB[row_ + N[0] * c] / AB[r + N[0] * c];
		AB[row_ + N[0] * c] -= AB[r + N[0] * c] * rowDivisor;
	}
}


int main() {

	int n, m, k;
	scanf("%d%d%d", &n, &m, &k);

	hipMemcpyToSymbol(HIP_SYMBOL(N), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(M), &m, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(K), &k, sizeof(int));

	int nm, nk;
	nm = n * m;
	nk = n * k;

	host_vector<double> matrAB_host(nm + nk);

	device_vector<double> matrAB_dev;

	srand(time(NULL));

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			//double rNum = (double)((rand() * rand()) % 1000000) / 2783;
			//cin >> matrAB_host[i + j * n];
			scanf("%lf", &matrAB_host[i + j * n]);
			//matrAB_host[i + j * n] = rNum;
		}
	}

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < k; ++j) {
			double rNum = (double)((rand() * rand()) % 1000000) / 5466;
			//cin >> matrAB_host[i + j * n + nm];
			scanf("%lf", &matrAB_host[i + j * n + nm]);
			//matrAB_host[i + j * n + nm] = rNum;
		}
	}


	//cerr << "N: " << n << " M: " << m << " K: " << k << "\n";


	//to GPU memory
	matrAB_dev = matrAB_host;



	// pointers to mem:
	double* AB_ptr = thrust::raw_pointer_cast(matrAB_dev.data());


	dim3 gridSz(BLOCK2, BLOCK2);
	dim3 blockSz(THREAD2, THREAD2);

	int row = 0; //i - col


	host_vector<int> indices(0);
	host_vector<int> indices2(0);


	hipEvent_t start, stop;
	float gpu_time = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);



	for (int i = 0; i < m && row < n; ++i) {

		auto iter = matrAB_dev.begin() + i * n;


		auto max_el = thrust::max_element(
			iter + row, iter + n, abs_comp()
		);


		int max_id = max_el - iter;


		if (fabs(*max_el) <= 1e-7) {
			continue;
		}
		else {
			if (max_id != row) {
				fastSwap << <BLOCK, THREAD >> > (row, i, max_id, AB_ptr);
				CSC(hipDeviceSynchronize());
			}
		}



		triagStep << <gridSz, blockSz >> > (AB_ptr, row, i);


		CSC(hipDeviceSynchronize());

		triagKill << <BLOCK, THREAD >> > (AB_ptr, row, i);

		CSC(hipDeviceSynchronize());


		indices.push_back(i);
		indices2.push_back(row);
		++row;

	}

	matrAB_host = matrAB_dev;




	host_vector<bool> i_bool(m, false);
	device_vector<bool> i_bool_dev(m);

	int freeX = indices.size();

	for (int i = 0; i < freeX; ++i) {
		i_bool[indices[i]] = true;
	}


	i_bool_dev = i_bool;

	matrAB_dev = matrAB_host;

	AB_ptr = thrust::raw_pointer_cast(matrAB_dev.data());


	// reversing step
	for (int i = 0; i < freeX - 1; ++i) {
		int curRow = indices2[freeX - i - 1];
		int curCol = indices[freeX - i - 1];


		backStep << <gridSz, blockSz >> > (AB_ptr, curRow, curCol);

		//CSC(hipDeviceSynchronize());

		backKill << <BLOCK, THREAD >> > (AB_ptr, curRow, curCol);

		CSC(hipDeviceSynchronize());
	}


	device_vector<int> indices_dev;
	device_vector<int> indices2_dev;


	host_vector<double> diags(freeX);
	device_vector<double> diags_dev;

	matrAB_host = matrAB_dev;

	for (int i = 0; i < freeX; ++i) {
		diags[i] = matrAB_host[indices2[i] + n * indices[i]];
	}

	indices_dev = indices;
	indices2_dev = indices2;
	diags_dev = diags;

	int* i_ptr = thrust::raw_pointer_cast(indices_dev.data());
	int* i2_ptr = thrust::raw_pointer_cast(indices2_dev.data());
	double* d_ptr = thrust::raw_pointer_cast(diags_dev.data());

	//divide rights

	division << <gridSz, blockSz >> > (AB_ptr, d_ptr, i_ptr, i2_ptr, freeX);

	CSC(hipDeviceSynchronize());

	matrAB_host = matrAB_dev;


	//AtoE
	for (int i = 0; i < freeX; ++i) {
		int curRow = indices2[i];
		int curCol = indices[i];

		matrAB_host[curRow + curCol * n] = 1;
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);

	// open log:
	ofstream log("logs.log", ios::app);
	log << "F" << endl;
	// threads
	log << BLOCK << "x" << THREAD << endl;
	// size:
	log << n << "x" << m << "x" << k << endl;
	// time:
	log << gpu_time << endl;
	log.close();





	int t = 0;
	//filling XMatrix
	
	for (int i = 0; i < m; ++i) {
		int curXRow;
		if (indices.size() > t) {
			curXRow = indices[t];
			if (i == curXRow) {
				for (int j = 0; j < k; ++j) {
					printf("%.10e ", matrAB_host[t + n * j + n * m]);
				}
				printf("\n");
				t += 1;
			}
			else {
				for (int j = 0; j < k; ++j) {
					printf("%.10e ", 0.0);
				}
				printf("\n");
			}
		}
		else {
			for (int j = 0; j < k; ++j) {
				printf("%.10e ", 0.0);
			}
			printf("\n");
		}

	}


	return 0;
}
