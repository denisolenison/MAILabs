#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
//for __syncthreads()
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)

#include <hip/device_functions.h>

#define SHARED_MEMORY 512
#define MAXIMAL_INTEGER 2147483647 // 2^31 - 1

using namespace std;


#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}															    \
} while (0)	

using namespace std;


class ClassSort {
public:
	int* intArray;
	int size = 0;

	ClassSort()
	{
		freopen(NULL, "rb", stdin);
		fread(&size, sizeof(int), 1, stdin);
		cerr << "Len: " << size << "\n";

		intArray = (int*)malloc(sizeof(*intArray) * size);

		fread(intArray, sizeof(int), size, stdin);

		//for (int i = 0; i < size; i++) {
		//	cerr << (int)intArray[i] << " ";
		//}

		fclose(stdin);
	};

	ClassSort(string inpFile) {
		FILE* file;
		if ((file = fopen(inpFile.c_str(), "rb")) == NULL) {
			cerr << "Can't load file" << "\n";
			exit(1);
		}

		fread(&size, sizeof(size), 1, file);
		cerr << "Len: " << size << "\n";
		intArray = (int*)malloc(sizeof(*intArray) * size);

		fread(intArray, sizeof(int), size, file);

		fclose(file);
	}

	void PrintRSS() {
		for (int i = 0; i < size - 1; i++) {
			if ((int)intArray[i + 1] < (int)intArray[i]) {
				cerr << (int)intArray[i] << " : " << (int)intArray[i+1] << "| ";
			}
		}
		cerr << "\n";
		freopen(NULL, "wb", stdout);
		fwrite(intArray, sizeof(*intArray), size, stdout);
		fclose(stdout);
	}

	void PrintRSS(string outFile) {
		for (int i = 0; i < size; i++) {
			cerr << (int)intArray[i] << " ";
		}
		cerr << "\n";

		FILE* file = fopen(outFile.c_str(), "wb");
		fwrite(intArray, sizeof(*intArray), size, file);
		fclose(file);
	}

	~ClassSort() {
		fclose(stdin);
		fclose(stdout);
		free(intArray);
	};
};


__device__ void oddEvenOne(const int tid, int* tmp, int shift, int len) {
	if ((tid + shift) % 2 == 0) {
		int a = tid;
		int b = tid + 1;
		if (b < len && tmp[a] > tmp[b]) {
			int temp = tmp[a];
			tmp[a] = tmp[b];
			tmp[b] = temp;
		}
	}
}


__global__ void OddEvenBlocks(int* theArray, int size, int shift2) {

	int tx = gridDim.x * blockDim.x;
	int tt = blockDim.x;
	int tid = threadIdx.x;
	int bid = blockDim.x * blockIdx.x;

	int tid_full = bid + threadIdx.x;

	__shared__ int tmp[SHARED_MEMORY];
	__shared__ int shift;


	for (int begin = bid + shift2; begin < size; begin += tx) {

		tid_full = begin + threadIdx.x;

		int end = (size > begin + tt ? begin + tt : size);
		int len = end - begin;

		if (tid < len) {
			tmp[tid] = theArray[tid_full];
		}
		else {
			tmp[tid] = MAXIMAL_INTEGER;
		}

		__syncthreads();

		for (int j = 0; j < len; ++j) {
			if (j % 2 == 0) {
				shift = 0;
			}
			else {
				shift = 1;
			}
			oddEvenOne(tid, tmp, shift, len);
			__syncthreads();
		}

		if (tid < len) {
			theArray[tid_full] = tmp[tid];
		}

		__syncthreads();
	}
}



__global__ void BitonicMerge(int* theArray, int size, int shift2) {


	int tx = gridDim.x * blockDim.x;
	int tt = blockDim.x; // 2 * SHARED_MEMORY
	int tid = threadIdx.x;
	int bid = blockDim.x * blockIdx.x;

	int shiftt = shift2;

	int tid_full = bid + threadIdx.x;
	__shared__ int tmp[2 * SHARED_MEMORY];



	for (int begin = bid + shiftt; begin < size; begin += tx) {
		tid_full = begin + threadIdx.x;

		int end = (size > begin + tt ? begin + tt : size);
		int len = end - begin;

		if ((tid < tt/2) && (tid < len)) {
			tmp[tid] = theArray[tid_full];
		}
		else if (tid < len) {
			tmp[tt + tt/2 - tid - 1] = theArray[tid_full];
		}
		else if ((tid < tt) && (tid >= tt / 2)) {
			tmp[tt + tt / 2 - tid - 1] = MAXIMAL_INTEGER;
		}
		else {
			tmp[tid] = MAXIMAL_INTEGER;
		}

		__syncthreads();

		int base = tt / 2;
		int shift = 0;
		int n_tid = tid;

		while (base >= 1) {
			if (n_tid >= base) {
				int opTid = n_tid - base;
				if (tmp[opTid + shift] > tmp[n_tid + shift]) {
					int temp = tmp[opTid + shift];
					tmp[opTid + shift] = tmp[n_tid + shift];
					tmp[n_tid + shift] = temp;
				}
				if (base >= 1) {
					base = base / 2;
					n_tid = n_tid - base;
					shift = shift + base;
				}
			}
			else {
				if (base >= 1) {
					base = base / 2;
				}
			}
			__syncthreads();
		}

		__syncthreads();


		if (tid < len) {
			theArray[tid_full] = tmp[tid];
		}

		__syncthreads();

	}
}


void Sorting(ClassSort* sort) {
	int* D_theArray;

	CSC(hipMalloc((void**)& D_theArray, sizeof(*D_theArray) * sort->size));

	CSC(hipMemcpy(D_theArray, sort->intArray, sizeof(*sort->intArray) * sort->size, hipMemcpyHostToDevice));

	//pre-entry odd-even sort

	OddEvenBlocks << <1024, SHARED_MEMORY >> > (D_theArray, sort->size, 0);

	CSC(hipGetLastError());


	//cycle bitonic-merge / odd-even sort
	int iters = (sort->size + 1) / SHARED_MEMORY + 2;
	for (int i = 0; i < iters; ++i) {
		int evOdd = i % 2;


		BitonicMerge << <1024, 2 * SHARED_MEMORY >> > (D_theArray, sort->size, evOdd * SHARED_MEMORY);
		CSC(hipGetLastError());


		//cerr << "Iter: " << i << "\n";
	}

	CSC(hipMemcpy(sort->intArray, D_theArray, sizeof(int) * sort->size, hipMemcpyDeviceToHost));
	CSC(hipFree(D_theArray));
}




int main(void) {

	//ClassSort theSort = ClassSort("test.data");
	//Sorting(&theSort);
	//theSort.PrintRSS("test_out.data");

	ClassSort theSort = ClassSort();
	Sorting(&theSort);
	theSort.PrintRSS();

	return 0;
}