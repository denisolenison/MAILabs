#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <mpi.h>
#include <string>
#include <algorithm>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}															    \
} while (0)	

int p1, p2, p3;
int g1, g2, g3;

// Index inside the block
#define _i(i, j, k) ((k + 1) * ((g2 + 2) * (g1 + 2)) + (j + 1) * (g1 + 2) + i + 1)
#define _ix(id) (((id) % (g1 + 2)) - 1)
#define _iy(id) ((((id) % ((g1 + 2) * (g2 + 2))) / (g1 + 2)) - 1)
#define _iz(id) (((id) / ((g1 + 2)*(g2 + 2))) - 1)

// Index by processes
#define _ib(i, j, k) ((k) * (p1 * p2) + (j) * p1 + (i))
#define _ibx(id) ((id) % p1)
#define _iby(id) (((id) % (p1 * p2)) / p1)
#define _ibz(id) ((id) / (p1 * p2))


#define printf(...) fprintf(File, __VA_ARGS__)



__global__ void ArrInits(double* fastData, double* fastNext, int g1, int g2, int g3, double U0) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int tZ = blockIdx.z * blockDim.z + threadIdx.z;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	int offsetZ = gridDim.z * blockDim.z;

	for (int a = tX - 1; a < g1 + 1; a += offsetX) {
		for (int b = tY - 1; b < g2 + 1; b += offsetY) {
			for (int c = tZ - 1; c < g3 + 1; c += offsetZ) {
				fastData[_i(a, b, c)] = U0;
				fastNext[_i(a, b, c)] = U0;
			}
		}
	}

}


//1 - left , 2 - right, 3 - up, 4 - down, 5 - front, 6 - back
#define LEFT 1
#define RIGHT 2
#define UP 3
#define DOWN 4
#define FRONT 5
#define BACK 6


__global__ void cuda_get_side(double* fastData, double* fastSide, int side, int g1, int g2, int g3) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	if (side == LEFT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastSide[j + k * g2] = fastData[_i(0, j, k)];
			}
		}
	}
	else if (side == RIGHT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastSide[j + k * g2] = fastData[_i(g1 - 1, j, k)];
			}
		}
	}
	else if (side == UP) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastSide[i + k * g1] = fastData[_i(i, 0, k)];
			}
		}
	}
	else if (side == DOWN) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastSide[i + k * g1] = fastData[_i(i, g2 - 1, k)];
			}
		}
	}
	else if (side == FRONT) {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastSide[i + j * g1] = fastData[_i(i, j, 0)];
			}
		}
	}
	else {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastSide[i + j * g1] = fastData[_i(i, j, g3 - 1)];
			}
		}
	}
}


__global__ void cuda_set_side(double* fastData, double* fastSide, int side, int g1, int g2, int g3) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	if (side == LEFT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastData[_i(-1, j, k)] = fastSide[j + k * g2];
			}
		}
	}
	else if (side == RIGHT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastData[_i(g1, j, k)] = fastSide[j + k * g2];
			}
		}
	}
	else if (side == UP) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, -1, k)] = fastSide[i + k * g1];
			}
		}
	}
	else if (side == DOWN) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, g2, k)] = fastSide[i + k * g1];
			}
		}
	}
	else if (side == FRONT) {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, j, -1)] = fastSide[i + j * g1];
			}
		}
	}
	else {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, j, g3)] = fastSide[i + j * g1];
			}
		}
	}
}


__global__ void cuda_side_edge_values(double* fastData, int side, int g1, int g2, int g3, double value) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	if (side == LEFT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastData[_i(-1, j, k)] = value;
			}
		}
	}
	else if (side == RIGHT) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int j = tY; j < g2; j += offsetY) {
				fastData[_i(g1, j, k)] = value;
			}
		}
	}
	else if (side == UP) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, -1, k)] = value;
			}
		}
	}
	else if (side == DOWN) {
		for (int k = tX; k < g3; k += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, g2, k)] = value;
			}
		}
	}
	else if (side == FRONT) {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, j, -1)] = value;
			}
		}
	}
	else {
		for (int j = tX; j < g2; j += offsetX) {
			for (int i = tY; i < g1; i += offsetY) {
				fastData[_i(i, j, g3)] = value;
			}
		}
	}
}



__global__ void cuda_main_function(double* fastData, double* fastNext, int g1,
 int g2, int g3, double hx, double hy, double hz) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int tZ = blockIdx.z * blockDim.z + threadIdx.z;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	int offsetZ = gridDim.z * blockDim.z;

	for (int k = tZ; k < g3; k += offsetZ) {
		for (int j = tY; j < g2; j += offsetY) {
			for (int i = tX; i < g1; i += offsetX) {
				fastNext[_i(i, j, k)] = 0.5 * ((fastData[_i(i + 1, j, k)] + 
					fastData[_i(i - 1, j, k)]) / (hx * hx) + (fastData[_i(i, j + 1, k)]
					+ fastData[_i(i, j - 1, k)]) / (hy * hy) + (fastData[_i(i, j, k + 1)]
					+ fastData[_i(i, j, k - 1)]) / (hz * hz)) /
					(1.0 / (hx * hx) + 1.0 / (hy * hy) + 1.0 / (hz * hz));
			}
		}
	}
}

__global__ void cuda_error_function(double* fastData, double* fastNext, int g1, int g2, int g3) {
	int tX = blockIdx.x * blockDim.x + threadIdx.x;
	int tY = blockIdx.y * blockDim.y + threadIdx.y;
	int tZ = blockIdx.z * blockDim.z + threadIdx.z;
	int offsetX = gridDim.x * blockDim.x;
	int offsetY = gridDim.y * blockDim.y;
	int offsetZ = gridDim.z * blockDim.z;

	for (int k = tZ - 1; k < g3 + 1; k += offsetZ) {
		for (int j = tY - 1; j < g2 + 1; j += offsetY) {
			for (int i = tX - 1; i < g1 + 1; i += offsetX) {
				bool lolkekval = (i != -1 && j != -1 && k != -1) *
					(i != g1 && j != g2 && k != g3);
				fastData[_i(i, j, k)] = fabs(fastNext[_i(i, j, k)] - fastData[_i(i, j, k)]) * lolkekval;
			}
		}
	}
}


int main(int argc, char** argv) {
	std::ios::sync_with_stdio(false);
	string outFile;
	int fileNameL = 16;

	int deviceCount;
	hipGetDeviceCount(&deviceCount);


	int id;
	int ib, jb, kb;
	int i, j, k, iter;
	int numproc, proc_name_len;
	int L = 16;

	char proc_name[MPI_MAX_PROCESSOR_NAME];

	double eps;
	double lx, ly, lz;
	double hx, hy, hz;
	double Udown, Uup, Uleft, Uright, Ufront, Uback;
	double U0;
	double * temp;

	MPI_Status status;

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Get_processor_name(proc_name, &proc_name_len);


	hipSetDevice(id % deviceCount);

	if (id == 0) {
		cerr << "Found " << deviceCount << " devices\n";
	}


	MPI_Barrier(MPI_COMM_WORLD);

	string nullString = "";
	const char* outFileC = nullString.c_str();



	//input data for 0 process
	if (id == 0) {

		cin >> p1 >> p2 >> p3;
		cin >> g1 >> g2 >> g3;
		cin >> outFile;
		cin >> eps;
		cin >> lx >> ly >> lz;
		cin >> Ufront >> Uback >> Uleft >> Uright >> Uup >> Udown;
		cin >> U0;

		outFileC = outFile.c_str();
		fileNameL = strlen(outFileC) + 1;
	}


	MPI_Barrier(MPI_COMM_WORLD);

	//send data to all processes
	MPI_Bcast(&p1, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&p2, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&p3, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&g1, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&g2, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&g3, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lz, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&Udown, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Uup, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Uleft, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Uright, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Ufront, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Uback, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&U0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&fileNameL, 1, MPI_INT, 0, MPI_COMM_WORLD);





	if (p1 * p2 * p3 != numproc) {
		MPI_Finalize();
		if (id == 0) {
			cout << "ERROR: proc.grid != processes\n";
		}

		return -1;
	}



	//block id by coordinates
	ib = _ibx(id);
	jb = _iby(id);
	kb = _ibz(id);

	iter = 0;

	//find hs
	hx = lx / ((double)p1 * (double)g1);
	hy = ly / ((double)p2 * (double)g2);
	hz = lz / ((double)p3 * (double)g3);

	char* buff = (char*)malloc(sizeof(char) * (g1 * g2 * g3 * p1 * p2 * p3 * L));


	double* fastData, * fastNext;

	double* fastLeftIn, * fastRightIn, * fastUPIn, * fastDownIn, * fastFrontIn, * fastBackIn;
	double* fastLeftOut, * fastRightOut, * fastUPOut, * fastDownOut, * fastFrontOut, * fastBackOut;

	double* LeftIn, * LeftOut, * RightIn, * RightOut;
	double* UPIn, * UPOut, * DownIn, * DownOut;
	double* FrontIn, * FrontOut, * BackIn, * BackOut;

	LeftIn = (double*)malloc(sizeof(double) * g2 * g3); LeftOut = (double*)malloc(sizeof(double) * g2 * g3);
	RightIn = (double*)malloc(sizeof(double) * g2 * g3); RightOut = (double*)malloc(sizeof(double) * g2 * g3);

	UPIn = (double*)malloc(sizeof(double) * g1 * g3); UPOut = (double*)malloc(sizeof(double) * g1 * g3);
	DownIn = (double*)malloc(sizeof(double) * g1 * g3); DownOut = (double*)malloc(sizeof(double) * g1 * g3);

	FrontIn = (double*)malloc(sizeof(double) * g1 * g2); FrontOut = (double*)malloc(sizeof(double) * g1 * g2);
	BackIn = (double*)malloc(sizeof(double) * g1 * g2); BackOut = (double*)malloc(sizeof(double) * g1 * g2);


	//Init GPU Memory
	CSC(hipMalloc((void**)&fastData, sizeof(double)* (g1 + 2)* (g2 + 2)* (g3 + 2)));
	CSC(hipMalloc((void**)&fastNext, sizeof(double)* (g1 + 2)* (g2 + 2)* (g3 + 2)));

	CSC(hipMalloc((void**)& fastLeftIn, sizeof(double)* g2 * g3));
	CSC(hipMalloc((void**)& fastRightIn, sizeof(double)* g2* g3));

	CSC(hipMalloc((void**)& fastUPIn, sizeof(double)* g1* g3));
	CSC(hipMalloc((void**)& fastDownIn, sizeof(double)* g1* g3));

	CSC(hipMalloc((void**)& fastFrontIn, sizeof(double)* g1* g2));
	CSC(hipMalloc((void**)& fastBackIn, sizeof(double)* g1* g2));

	CSC(hipMalloc((void**)& fastLeftOut, sizeof(double)* g2* g3));
	CSC(hipMalloc((void**)& fastRightOut, sizeof(double)* g2* g3));

	CSC(hipMalloc((void**)& fastUPOut, sizeof(double)* g1* g3));
	CSC(hipMalloc((void**)& fastDownOut, sizeof(double)* g1* g3));

	CSC(hipMalloc((void**)& fastFrontOut, sizeof(double)* g1* g2));
	CSC(hipMalloc((void**)& fastBackOut, sizeof(double)* g1* g2));

	int SIZE = 4;

	dim3 gridSz(SIZE, SIZE, SIZE);
	dim3 blockSz(SIZE, SIZE, SIZE);

	ArrInits << < gridSz, blockSz >> > (fastData, fastNext, g1, g2, g3, U0);

	//make buffer

	int buffer_size;

	MPI_Pack_size((g1 + 2) * (g2 + 2) * (g3 + 2), MPI_DOUBLE, MPI_COMM_WORLD, &buffer_size);

	buffer_size = 2 * (buffer_size + MPI_BSEND_OVERHEAD); //6 edges


	double* buffer = (double*)malloc(buffer_size);

	MPI_Buffer_attach(buffer, buffer_size);

	//block init

	//requests

	MPI_Request send_request1_1, recv_request1_1;
	MPI_Request send_request2_1, recv_request2_1;
	MPI_Request send_request3_1, recv_request3_1;

	MPI_Request send_request1_2, recv_request1_2;
	MPI_Request send_request2_2, recv_request2_2;
	MPI_Request send_request3_2, recv_request3_2;


	double* errors;
	errors = (double*)malloc(numproc * sizeof(double));

	//string debug_name = "process_debug" + to_string(id) + ".txt";

	int size2 = 8;

	dim3 blocks(size2, size2);
	dim3 threads(size2, size2);

	double maxErr = 0;
	do {
		//send and get data
		MPI_Barrier(MPI_COMM_WORLD);


		if (ib > 0) { //only left side
			cuda_get_side << < blocks, threads >> > (fastData, fastLeftIn, LEFT, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(LeftIn, fastLeftIn, sizeof(double)* g2* g3, hipMemcpyDeviceToHost));
			MPI_Isend(LeftIn, g2* g3, MPI_DOUBLE, _ib(ib - 1, jb, kb), 0, MPI_COMM_WORLD, &send_request1_1);
		}

		if (jb > 0) { //only up side
			cuda_get_side << < blocks, threads >> > (fastData, fastUPIn, UP, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(UPIn, fastUPIn, sizeof(double)* g1* g3, hipMemcpyDeviceToHost));
			MPI_Isend(UPIn, g1* g3, MPI_DOUBLE, _ib(ib, jb - 1, kb), 0, MPI_COMM_WORLD, &send_request2_2);
		}

		if (kb > 0) { //only front side
			cuda_get_side << < blocks, threads >> > (fastData, fastFrontIn, FRONT, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(FrontIn, fastFrontIn, sizeof(double)* g1* g2, hipMemcpyDeviceToHost));

			MPI_Isend(FrontIn, g1* g2, MPI_DOUBLE, _ib(ib, jb, kb - 1), 0, MPI_COMM_WORLD, &send_request3_2);
		}


		//wait for data

		if (ib > 0) { //only left side
			MPI_Wait(&send_request1_1, &status);
		}

		if (jb > 0) { //only up side
			MPI_Wait(&send_request2_2, &status);
		}

		if (kb > 0) { //only front side
			MPI_Wait(&send_request3_2, &status);
		}



		//set new data

		if (ib + 1 < p1) { //get right side
			MPI_Irecv(RightOut, g2 * g3, MPI_DOUBLE, _ib(ib + 1, jb, kb), 0, MPI_COMM_WORLD, &recv_request1_2);
			MPI_Wait(&recv_request1_2, &status);

			CSC(hipMemcpy(fastRightOut, RightOut, sizeof(double)* g2* g3, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastRightOut, RIGHT, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, RIGHT, g1, g2, g3, Uright);
			CSC(hipGetLastError());
		}

		if (jb + 1 < p2) { //get down side
			MPI_Irecv(DownOut, g1* g3, MPI_DOUBLE, _ib(ib, jb + 1, kb), 0, MPI_COMM_WORLD, &recv_request2_1);
			MPI_Wait(&recv_request2_1, &status);

			CSC(hipMemcpy(fastDownOut, DownOut, sizeof(double)* g1* g3, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastDownOut, DOWN, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, DOWN, g1, g2, g3, Udown);
			CSC(hipGetLastError());
		}


		if (kb + 1 < p3) { //get back side
			MPI_Irecv(BackOut, g1* g2, MPI_DOUBLE, _ib(ib, jb, kb + 1), 0, MPI_COMM_WORLD, &recv_request3_1);
			MPI_Wait(&recv_request3_1, &status);

			CSC(hipMemcpy(fastBackOut, BackOut, sizeof(double)* g1* g2, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastBackOut, BACK, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, BACK, g1, g2, g3, Uback);
			CSC(hipGetLastError());
		}





		if (ib + 1 < p1) { //only right side
			cuda_get_side << < blocks, threads >> > (fastData, fastRightIn, RIGHT, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(RightIn, fastRightIn, sizeof(double) * g2 * g3, hipMemcpyDeviceToHost));
			MPI_Isend(RightIn, g2 * g3, MPI_DOUBLE, _ib(ib + 1, jb, kb), 0, MPI_COMM_WORLD, &send_request1_2);
		}

		if (jb + 1 < p2) { //only down side
			cuda_get_side << < blocks, threads >> > (fastData, fastDownIn, DOWN, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(DownIn, fastDownIn, sizeof(double) * g1 * g3, hipMemcpyDeviceToHost));
			MPI_Isend(DownIn, g1 * g3, MPI_DOUBLE, _ib(ib, jb + 1, kb), 0, MPI_COMM_WORLD, &send_request2_1);
		}

		if (kb + 1 < p3) { //only back side
			cuda_get_side << < blocks, threads >> > (fastData, fastBackIn, BACK, g1, g2, g3);
			CSC(hipGetLastError());
			CSC(hipMemcpy(BackIn, fastBackIn, sizeof(double) * g1 * g2, hipMemcpyDeviceToHost));

			MPI_Isend(BackIn, g1 * g2, MPI_DOUBLE, _ib(ib, jb, kb + 1), 0, MPI_COMM_WORLD, &send_request3_1);
		}

		if (ib + 1 < p1) { //only right side
			MPI_Wait(&send_request1_2, &status);
		}
		if (jb + 1 < p2) { //only down side
			MPI_Wait(&send_request2_1, &status);
		}
		if (kb + 1 < p3) { //only back side
			MPI_Wait(&send_request3_1, &status);
		}



		if (ib > 0) { //get left side
			MPI_Irecv(LeftOut, g2* g3, MPI_DOUBLE, _ib(ib - 1, jb, kb), 0, MPI_COMM_WORLD, &recv_request1_1);
			MPI_Wait(&recv_request1_1, &status);

			CSC(hipMemcpy(fastLeftOut, LeftOut, sizeof(double)* g2* g3, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastLeftOut, LEFT, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, LEFT, g1, g2, g3, Uleft);
			CSC(hipGetLastError());
		}


		if (jb > 0) { //get up side
			MPI_Irecv(UPOut, g1* g3, MPI_DOUBLE, _ib(ib, jb - 1, kb), 0, MPI_COMM_WORLD, &recv_request2_2);
			MPI_Wait(&recv_request2_2, &status);

			CSC(hipMemcpy(fastUPOut, UPOut, sizeof(double)* g1* g3, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastUPOut, UP, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, UP, g1, g2, g3, Uup);
			CSC(hipGetLastError());
		}


		if (kb > 0) { //get front side
			MPI_Irecv(FrontOut, g1* g2, MPI_DOUBLE, _ib(ib, jb, kb - 1), 0, MPI_COMM_WORLD, &recv_request3_2);
			MPI_Wait(&recv_request3_2, &status);

			CSC(hipMemcpy(fastFrontOut, FrontOut, sizeof(double)* g1* g2, hipMemcpyHostToDevice));
			cuda_set_side << < blocks, threads >> > (fastData, fastFrontOut, FRONT, g1, g2, g3);
			CSC(hipGetLastError());
		}
		else {
			cuda_side_edge_values << < blocks, threads >> > (fastData, FRONT, g1, g2, g3, Ufront);
			CSC(hipGetLastError());
		}

		MPI_Barrier(MPI_COMM_WORLD);
		
		cuda_main_function << < gridSz, blockSz >> > (fastData, fastNext, g1, g2, g3, hx, hy, hz);

		cuda_error_function << < gridSz, blockSz >> > (fastData, fastNext, g1, g2, g3);

		thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(fastData);
		
		double epsTemp[1];

		epsTemp[0] = *(thrust::max_element(d_ptr, d_ptr + (g1 + 2)*(g2 + 2)*(g3+2)));


		MPI_Barrier(MPI_COMM_WORLD);


		MPI_Allgather(epsTemp, 1, MPI_DOUBLE, errors, 1, MPI_DOUBLE, MPI_COMM_WORLD);
		epsTemp[0] = 0;
		for (i = 0; i < numproc; ++i) {
			epsTemp[0] = max(epsTemp[0], errors[i]);
		}

		temp = fastNext;
		fastNext = fastData;
		fastData = temp;

		maxErr = epsTemp[0];

		iter += 1;

	} while (maxErr >= eps);


	//cout << iter << "\n";

	CSC(hipFree(fastNext));

	double* data = (double*)malloc(sizeof(double) * (g1+2)*(g2+2)*(g3+2));

	CSC(hipMemcpy(data, fastData, sizeof(double)*(g1+2)*(g2+2)*(g3+2), hipMemcpyDeviceToHost));


	CSC(hipFree(fastData));

	MPI_Barrier(MPI_COMM_WORLD);

	memset(buff, ' ', L * g1 * g2 * g3 * sizeof(char));

	for (k = 0; k < g3; ++k) {
		for (j = 0; j < g2; j++) {
			for (i = 0; i < g1; i++) {
				if ((i == 0 && j == 0 && (k != 0 || _iby(id) != 0 || _ibz(id) != 0)) && _ibx(id) == 0 && (_iby(id) > 0 || _ibz(id) > 0 || k > 0)) {
					sprintf(buff + (k * (g1 * g2) + j * g1 + i) * L, "\n\n%.6e ", data[_i(i, j, k)]);
				}
				else if (i == 0 && _ibx(id) == 0 && (j > 0 || k > 0)) {
					sprintf(buff + (k * (g1 * g2) + j * g1 + i) * L, "\n%.6e ", data[_i(i, j, k)]);
				}
				else {
					sprintf(buff + (k * (g1 * g2) + j * g1 + i) * L, "%.6e ", data[_i(i, j, k)]);
				}
			}
		}
	}
	for (i = 0; i < g1 * g2 * g3 * L; ++i) {
		if (buff[i] == '\0')
			buff[i] = ' ';
		//cout << buff[i];
	}



	MPI_Datatype fileCube;

	MPI_Datatype fileType;

	int count = g1 * g2 * g3;


	int* lens = new int[count];

	for (int i = 0; i < count; ++i) lens[i] = L;

	MPI_Aint* adrToFile = new MPI_Aint[count];

	MPI_Aint* adrInFile = new MPI_Aint[count];

	for (int k = 0; k < g3; ++k) {
		for (int j = 0; j < g2; ++j) {
			for (int i = 0; i < g1; ++i) {
				int CZ = g1 * g2 * g3;
				adrToFile[k * g2 * g1 + j * g1 + i] = (((_ibz(id) * p1 * p2 * CZ) + (k * p1 * g1 * p2 * g2)) +
					((_iby(id) * p1 * g1 * g2) + (j * p1 * g1)) + (_ibx(id) * g1 + i)) * sizeof(char) * L;
			}
		}
	}

	for (int k = 0; k < g3; ++k) {
		for (int j = 0; j < g2; ++j) {
			for (int i = 0; i < g1; ++i) {
				int t = k * g2 * g1 + j * g1 + i;
				adrInFile[t] = t * sizeof(char) * L;
			}
		}
	}


	MPI_Type_create_hindexed(count, lens, adrToFile, MPI_CHAR, &fileCube);
	MPI_Type_create_hindexed(count, lens, adrInFile, MPI_CHAR, &fileType);

	MPI_Type_commit(&fileCube);
	MPI_Type_commit(&fileType);


	if (id == 0) {
		cerr << "Process GRID: " << p1 << "x" << p2 << "x" << p3 << "\n";
		cerr << "Num GRID: " << g1 << "x" << g2 << "x" << g3 << "\n";
		cerr << "File name " << outFile << "\n";
		cerr << "Eps: " << eps << "\n";
		cerr << "lx: " << lx << " ly: " << ly << " lz: " << lz << "\n";
		cerr << "Us: " << Ufront << " , " << Uback << " , " << Uleft << " , " << Uright << " , " << Uup << " , " << Udown << "\n";
		cerr << "U0: " << U0 << "\n";
		cerr << "Iterations: " << iter << "\n";
	}

	MPI_File fp;

	MPI_Request* send_request = new MPI_Request[numproc - 1];
	MPI_Request* recv_request = new MPI_Request[numproc - 1];


	char* buffName = new char[fileNameL];

	if (id == 0) {
		for (int t = 1; t < numproc; ++t) {
			MPI_Isend(outFileC, fileNameL * sizeof(char), MPI_CHAR, t, 0, MPI_COMM_WORLD, &send_request[t - 1]);
			MPI_Wait(&send_request[t - 1], &status);
		}
		for (int x = 0; x < fileNameL; ++x) {
			buffName[x] = outFileC[x];;
		}
	}

	MPI_Barrier(MPI_COMM_WORLD);

	if (id != 0) {
		MPI_Irecv(buffName, fileNameL * sizeof(char), MPI_CHAR, 0, 0, MPI_COMM_WORLD, &recv_request[id - 1]);
		MPI_Wait(&recv_request[id - 1], &status);
	}

	MPI_Barrier(MPI_COMM_WORLD);



	MPI_File_open(MPI_COMM_WORLD, buffName, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);

	MPI_File_set_view(fp, 0, MPI_CHAR, fileCube, "native", MPI_INFO_NULL);

	MPI_File_write(fp, buff, 1, fileType, &status);

	MPI_File_close(&fp);


	MPI_Barrier(MPI_COMM_WORLD);


	MPI_Finalize();


	free(data);
	free(buffer);

	return 0;
}